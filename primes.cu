
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK 512

__global__ void isPrimeKernel(unsigned long long int number, bool *result) {
    __shared__ bool notPrime;
    if (threadIdx.x == 0) notPrime = false;
    __syncthreads();

    unsigned long long int index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long long int stride = blockDim.x * gridDim.x;

    if (number % 2 == 0) {
        *result = false;
        return;
    }

    for (unsigned long long int i = max(index * 2 + 3, static_cast<unsigned long long int>(3)); i <= sqrtf((double)number); i += 2 * stride) {
        if (notPrime) return;
        if (number % i == 0) {
            notPrime = true;
            *result = false;
            return;
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: ./primes number\n");
        return 1;
    }

    unsigned long long int number = atoll(argv[1]);
    //unsigned long long int number = 7;
    bool result = true;
    bool *dev_result;

    hipMalloc((void**)&dev_result, sizeof(bool));
    hipMemcpy(dev_result, &result, sizeof(bool), hipMemcpyHostToDevice);

    int blocks = 65535;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, NULL);

    isPrimeKernel<<<blocks, THREADS_PER_BLOCK>>>(number, dev_result);

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(&result, dev_result, sizeof(bool), hipMemcpyDeviceToHost);

    if (result) {
        printf("%llu is prime\n", number);
    } else {
        printf("%llu is not prime\n", number);
    }
    printf("%f\n", milliseconds);
    hipFree(dev_result);

    return 0;
}